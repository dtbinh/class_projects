
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int addition (int a, int b)
{
  int r;
  r=a+b;
  return r;
}

__global__ void device_greetings(void)
{
  printf("Hello, world from the device!\n");
  char n[] = "big!\n";
  printf(n);

  int z;
  z = addition(5,3);
  printf("The result is %d\n", z);
}

int main(void)
{
  // greet from the host
  printf("Hello, world from the host!\n");

  // launch a kernel with a single thread to greet from the device
  device_greetings<<<1,1>>>();
  hipDeviceSynchronize();


  return 0;
}