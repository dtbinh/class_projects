
#include <hip/hip_runtime.h>
#include <stdio.h>

#define SIZE	2048

__global__ void VectorAdd (int *a, int *b, int *c, int n)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (i < n)
		c[i] = a[i] + b[i];
}

int main()
{
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;

	a = (int *)malloc(SIZE*sizeof(int));
	b = (int *)malloc(SIZE*sizeof(int));
	c = (int *)malloc(SIZE*sizeof(int));

	hipMalloc( &d_a, SIZE*sizeof(int));
	hipMalloc( &d_b, SIZE*sizeof(int));
	hipMalloc( &d_c, SIZE*sizeof(int));

	for( int i = 0; i < SIZE; ++i )
	{
		a[i] = i;
		b[i] = i;
		c[i] = 0;
	}

	hipMemcpy( d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice );

	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(2048/threadsPerBlock.x);

	VectorAdd<<< numBlocks, threadsPerBlock >>>(d_a, d_b, d_c, SIZE);

	hipMemcpy( a, d_a, SIZE*sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( b, d_b, SIZE*sizeof(int), hipMemcpyDeviceToHost );
	hipMemcpy( c, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost );

	for( int i = 0; i < SIZE; ++i){
		printf("a[%d] = %d\n", i, a[i]);
		printf("b[%d] = %d\n", i, b[i]);
		printf("c[%d] = %d\n", i, c[i]);
	}

	free(a);
	free(b);
	free(c);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}