#include "hip/hip_runtime.h"
/********************************
Based on code by:
Lorenzo Seidenari (sixmoney@virgilio.it)
*********************************/

#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctype.h>

#define MAX_SEQUENCE_LENGTH 100000

int  n; 
int  m;
int levenshtein_distance(char *s,char*t);
int minimum(int a,int b,int c);

//-----------------------------------------------------------------------------
void cleanString(char string[]) {
  //Removes all spaces from string pointed to by "string", converts characters
  //to uppercase, and deletes a terminating newline character.
	int i, current;
	int length = strlen(string);

	current = 0;
	for(i=0;i<length;i++) {
		if(string[i]=='\n') {
			string[current++] = '\0';
			break;
		}
		else if(string[i]!=' ') {
			string[current++] = toupper(string[i]);
		}
	}
}
//-----------------------------------------------------------------------------
int importFASTA(char *filename, char *sequence) {
  //Reads a file, located at path specified by "filename", containing a FASTA
  //sequence. It finds the first full, complete sequence in the file, stores
  //it in "sequence", and returns the length of the sequence, or -1 on failure.
	FILE *fastaFile;
	char input[256];
  int readFlag; //set to 1 once a sequence has been read in
  int length;

  //open the file
  if((fastaFile = fopen(filename, "r")) == NULL) {
  	return -1;
  }

  sequence[0] = '\0';

  //read the full first sequence, discarding unnecessary headers
  readFlag=0;
  length = 0;
  while(fgets(input,256,fastaFile)!=NULL) {
    //is it a header or a comment?
  	if(input[0]=='>' || input[0]==';') {
  		if(readFlag) break;
  		else continue;
  	}
  	else readFlag = 1;

  	cleanString(input);
  	length += strlen(input);

  	strncat(sequence,input,MAX_SEQUENCE_LENGTH-length - 1);
  }
  //Add a terminatng null character, just in case
  sequence[length] = '\0';

  fclose(fastaFile);
  return length;
}


/****************************************/
/*Implementation of Levenshtein distance*/
/****************************************/

__host__ void levenshtein_distance(char *s, char*t, int answer)
/*Compute levenshtein distance between s and t*/
{
	//Step 1
	int k,i,j,cost,*d,distance;
	if(n!=0&&m!=0)
	{
		d=(int *)malloc((sizeof(int))*(m+1)*(n+1));
		m++;
		n++;
    	//Step 2	
		for(k=0;k<n;k++)
			d[k]=k;
		for(k=0;k<m;k++)
			d[k*n]=k;
    	//Step 3 and 4	
		for(i=1;i<n;i++)
			for(j=1;j<m;j++)
			{
        	//Step 5
				if(s[i-1]==t[j-1])
					cost=0;
				else
					cost=1;
        		//Step 6			 
				d[j*n+i]=minimum(d[(j-1)*n+i]+1,d[j*n+i-1]+1,d[(j-1)*n+i-1]+cost);
			}
		distance=d[n*m-1];
		free(d);
		distance +=answer;
		//return distance;
	}
		
}

__device__	int minimum(int a,int b,int c)
/*Gets the minimum of three values*/
{
	int min=a;
	if(b<min)
		min=b;
	if(c<min)
		min=c;
	return min;
}

int main(int argc, char *argv[]) {
	char *d_a, *d_b;
	int *d_distance, *d_n, *d_m;
	int *distance = 0;
	char A[MAX_SEQUENCE_LENGTH+1];
	char B[MAX_SEQUENCE_LENGTH+1];

	if(argc < 3) {
		printf("Usage: new_edit_distance <sequence1> <sequence2>\n");
		printf("<sequence1>: file containing the first sequence, FASTA format\n");
		printf("<sequence2>: file containing the second sequence, FASTA format\n");
		return EXIT_FAILURE;
	}

	n = importFASTA(argv[1],A);
	m = importFASTA(argv[2],B);

	hipMalloc( (void**)&d_a, strlen(A) * sizeof(char) );
	hipMalloc( (void**)&d_b, strlen(B) * sizeof(char) );
	hipMalloc( &d_distance, sizeof(int) );
	hipMalloc( &d_n, sizeof(int) );
	hipMalloc( &d_m, sizeof(int) );

	hipMemcpy( d_a, A, strlen(A) * sizeof(char), hipMemcpyHostToDevice );
	hipMemcpy( d_b, B, strlen(B) * sizeof(char), hipMemcpyHostToDevice );
	hipMemcpy( d_distance, distance, sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_n, &n, sizeof(int), hipMemcpyHostToDevice );
	hipMemcpy( d_m, &m, sizeof(int), hipMemcpyHostToDevice );

	levenshtein_distance<<< 1, 1 >>>(d_a, d_b, *d_distance);

	//score = levenshtein_distance(A,B);
	hipMemcpy( distance, d_distance, sizeof(int), hipMemcpyDeviceToHost );

	printf("%d\n", *distance);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_distance);
	hipFree(d_n);
	hipFree(d_m);

	return EXIT_SUCCESS;
}
