/********************************
Based on code by:
Lorenzo Seidenari (sixmoney@virgilio.it)
*********************************/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctype.h>

#define MAX_SEQUENCE_LENGTH 100000

int  n; 
int  m;
int levenshtein_distance(char *s,char*t);
int minimum(int a,int b,int c);

//-----------------------------------------------------------------------------
void cleanString(char string[]) {
  //Removes all spaces from string pointed to by "string", converts characters
  //to uppercase, and deletes a terminating newline character.
  int i, current;
  int length = strlen(string);

  current = 0;
  for(i=0;i<length;i++) {
    if(string[i]=='\n') {
      string[current++] = '\0';
      break;
    }
    else if(string[i]!=' ') {
      string[current++] = toupper(string[i]);
    }
  }
}
//-----------------------------------------------------------------------------
int importFASTA(char *filename, char *sequence) {
  //Reads a file, located at path specified by "filename", containing a FASTA
  //sequence. It finds the first full, complete sequence in the file, stores
  //it in "sequence", and returns the length of the sequence, or -1 on failure.
  FILE *fastaFile;
  char input[256];
  int readFlag; //set to 1 once a sequence has been read in
  int length;

  //open the file
  if((fastaFile = fopen(filename, "r")) == NULL) {
    return -1;
  }

  sequence[0] = '\0';

  //read the full first sequence, discarding unnecessary headers
  readFlag=0;
  length = 0;
  while(fgets(input,256,fastaFile)!=NULL) {
    //is it a header or a comment?
    if(input[0]=='>' || input[0]==';') {
      if(readFlag) break;
      else continue;
    }
    else readFlag = 1;

    cleanString(input);
    length += strlen(input);

    strncat(sequence,input,MAX_SEQUENCE_LENGTH-length - 1);
  }
  //Add a terminatng null character, just in case
  sequence[length] = '\0';

  fclose(fastaFile);
  return length;
}


/****************************************/
/*Implementation of Levenshtein distance*/
/****************************************/

__device__ int levenshtein_distance(char *s,char*t, int one, int two)
/*Compute levenshtein distance between s and t*/
{
  //Step 1
  int k,i,j,cost,*d,distance;
  if(one!=0&&two!=0)
  {
  	printf ("%d\n", 1);
    d = (int *)malloc((sizeof(int))*(two+1)*(one+1));
    two++;
    one++;
    //Step 2
    for(k=0;k<one;k++){
        d[k]=k;
    }
    for(k=0;k<two;k++){
        d[k*one]=k;
    }
    //Step 3 and 4	
    for(i=1;i<one;i++)
      for(j=1;j<two;j++)
	{
        //Step 5
        if(s[i-1]==t[j-1])
          cost=0;
        else
          cost=1;
        //Step 6	
        int min = d[(j-1)*one+i]+1;
        if (d[j*one+i-1]+1 < min)
        	min = d[j*one+i-1]+1;
        if (d[(j-1)*one+i-1]+cost < min)
        	min = d[(j-1)*one+i-1]+cost;
        d[j*one+i] = min;
    }
    distance=d[one*two-1];
    printf ("%d\n", distance);
    free(d);
    return distance;
  }
  else 
    return -1; //a negative return value means that one or both strings are empty.
}

__global__ void kernel (int *score, char *s, char*t, int one, int two, int n)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < n-1){
		printf ("%d executed %d\n", i, n);
		score[i]=levenshtein_distance(s,t,one,two);
	}
    return;
}

int main(int argc, char *argv[]) {
  int *score = 0, *d_s = 0;
  char *d_A, *d_B;
  char A[MAX_SEQUENCE_LENGTH+1];
  char B[MAX_SEQUENCE_LENGTH+1];

  if(argc < 3) {
    printf("Usage: new_edit_distance <sequence1> <sequence2>\n");
    printf("<sequence1>: file containing the first sequence, FASTA format\n");
    printf("<sequence2>: file containing the second sequence, FASTA format\n");
    return EXIT_FAILURE;
  }

	int a = 2;
  	n = importFASTA(argv[1],A);
  	m = importFASTA(argv[2],B);

  	dim3 threadsPerBlock(1, 1);
	dim3 numBlocks(a/threadsPerBlock.x, a/threadsPerBlock.y);
	
  	hipMalloc(&d_A, MAX_SEQUENCE_LENGTH+1);
	hipMalloc(&d_B, MAX_SEQUENCE_LENGTH+1);

	hipMemcpy( d_s, score, sizeof(int)*(a), hipMemcpyHostToDevice );
    printf ("%s\n", hipGetErrorString(hipGetLastError()));
	score = (int *)malloc(sizeof(int)*a);

	kernel<<<numBlocks, threadsPerBlock>>>(d_s, d_A, d_B, n, m, a);
	printf ("%s\n", hipGetErrorString(hipGetLastError()));
	hipDeviceSynchronize();

	hipMemcpy( score, d_s, sizeof(int)*a, hipMemcpyDeviceToHost );
    printf ("%s\n", hipGetErrorString(hipGetLastError()));

	printf("%d\n", score[0]);

	hipFree(d_s);
	hipFree(d_A);
	hipFree(d_B);
	free(score);

  return EXIT_SUCCESS;
}
