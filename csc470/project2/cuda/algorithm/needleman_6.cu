/********************************
Based on code by:
Lorenzo Seidenari (sixmoney@virgilio.it)
*********************************/


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctype.h>

#define MAX_SEQUENCE_LENGTH 100000

int  n; 
int  m;
int levenshtein_distance(char *s,char*t);
int minimum(int a,int b,int c);

//-----------------------------------------------------------------------------
void cleanString(char string[]) {
  //Removes all spaces from string pointed to by "string", converts characters
  //to uppercase, and deletes a terminating newline character.
  int i, current;
  int length = strlen(string);

  current = 0;
  for(i=0;i<length;i++) {
    if(string[i]=='\n') {
      string[current++] = '\0';
      break;
    }
    else if(string[i]!=' ') {
      string[current++] = toupper(string[i]);
    }
  }
}
//-----------------------------------------------------------------------------
int importFASTA(char *filename, char *sequence) {
  //Reads a file, located at path specified by "filename", containing a FASTA
  //sequence. It finds the first full, complete sequence in the file, stores
  //it in "sequence", and returns the length of the sequence, or -1 on failure.
  FILE *fastaFile;
  char input[256];
  int readFlag; //set to 1 once a sequence has been read in
  int length;

  //open the file
  if((fastaFile = fopen(filename, "r")) == NULL) {
    return -1;
  }

  sequence[0] = '\0';

  //read the full first sequence, discarding unnecessary headers
  readFlag=0;
  length = 0;
  while(fgets(input,256,fastaFile)!=NULL) {
    //is it a header or a comment?
    if(input[0]=='>' || input[0]==';') {
      if(readFlag) break;
      else continue;
    }
    else readFlag = 1;

    cleanString(input);
    length += strlen(input);

    strncat(sequence,input,MAX_SEQUENCE_LENGTH-length - 1);
  }
  //Add a terminatng null character, just in case
  sequence[length] = '\0';

  fclose(fastaFile);
  return length;
}


/****************************************/
/*Implementation of Levenshtein distance*/
/****************************************/

__device__ int levenshtein_distance(char *s,char*t, int one, int two, int *d)
/*Compute levenshtein distance between s and t*/
{
  //Step 1
  int i,j,cost,distance;
  if(one!=0&&two!=0)
  {
  	printf ("%d\n", 1);
    two++;
    one++;
    //Step 3 and 4	
    for(i=1;i<one;i++)
      for(j=1;j<two;j++)
	{
        //Step 5
        if(s[i-1]==t[j-1])
          cost=0;
        else
          cost=1;
        //Step 6	
        int min = d[(j-1)*one+i]+1;
        if (d[j*one+i-1]+1 < min)
        	min = d[j*one+i-1]+1;
        if (d[(j-1)*one+i-1]+cost < min)
        	min = d[(j-1)*one+i-1]+cost;
        d[j*one+i] = min;
        //d[j*one+i]=minimum(d[(j-1)*one+i]+1,d[j*one+i-1]+1,d[(j-1)*one+i-1]+cost);
    }
    distance=d[one*two-1];
    printf ("%d\n", distance);
    free(d);
    return distance;
  }
  else 
    return -1; //a negative return value means that one or both strings are empty.
}

__global__ void kernel (int *score, char *s, char*t, int one, int two, int n, int *d)
{
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < n-1){
		printf ("%d executed %d\n", i, n);
		score[i]=levenshtein_distance(s,t,one,two,d);
	}
	else
		return;
}

int main(int argc, char *argv[]) {
  int *score = 0, *d_s = 0, *d = 0, *d_D = 0, k = 0;
  char *d_A, *d_B;
  char A[MAX_SEQUENCE_LENGTH+1];
  char B[MAX_SEQUENCE_LENGTH+1];

  if(argc < 3) {
    printf("Usage: new_edit_distance <sequence1> <sequence2>\n");
    printf("<sequence1>: file containing the first sequence, FASTA format\n");
    printf("<sequence2>: file containing the second sequence, FASTA format\n");
    return EXIT_FAILURE;
  }

	int a = 2;
  	n = importFASTA(argv[1],A);
  	m = importFASTA(argv[2],B);

  	hipMalloc(&d_D, (sizeof(int))*(m+1)*(n+1));

	d = (int *)malloc(sizeof(int)*(m+1)*(n+1));

	n++;
	m++;

	for(k=0;k<n;k++){
		d[k]=k;
	}
	for(k=0;k<m-1;k++){
		d[k*n]=k;
	}

	n--;
	m--;

	hipMemcpy( d_D, d, sizeof(int)*(m+1)*(n+1), hipMemcpyHostToDevice );

  	dim3 threadsPerBlock(1, 1);
	dim3 numBlocks(a/threadsPerBlock.x, a/threadsPerBlock.y);
	
  	hipMalloc(&d_A, MAX_SEQUENCE_LENGTH+1);
	hipMalloc(&d_B, MAX_SEQUENCE_LENGTH+1);

	hipMemcpy( d_s, score, sizeof(int)*a, hipMemcpyHostToDevice );
	score = (int *)malloc(sizeof(int)*a);

	kernel<<<numBlocks, threadsPerBlock>>>(d_s, d_A, d_B, n, m, a, d_D);
	printf ("%s\n", hipGetErrorString(hipGetLastError()));
	hipDeviceSynchronize();

	hipMemcpy( score, d_s, sizeof(int)*a, hipMemcpyDeviceToHost );

	printf("%d\n", score[0]);

	hipFree(d_s);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_D);
	free(d);
	free(score);

  return EXIT_SUCCESS;
}
