
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctype.h>

#define MAX_SEQUENCE_LENGTH 100000

int  n; 
int  m;
int levenshtein_distance(char *s,char*t);
int minimum(int a,int b,int c);

//-----------------------------------------------------------------------------
void cleanString(char string[]) {
  //Removes all spaces from string pointed to by "string", converts characters
  //to uppercase, and deletes a terminating newline character.
	int i, current;
	int length = strlen(string);

	current = 0;
	for(i=0;i<length;i++) {
		if(string[i]=='\n') {
			string[current++] = '\0';
			break;
		}
		else if(string[i]!=' ') {
			string[current++] = toupper(string[i]);
		}
	}
}
//-----------------------------------------------------------------------------
int importFASTA(char *filename, char *sequence) {
  //Reads a file, located at path specified by "filename", containing a FASTA
  //sequence. It finds the first full, complete sequence in the file, stores
  //it in "sequence", and returns the length of the sequence, or -1 on failure.
	FILE *fastaFile;
	char input[256];
  int readFlag; //set to 1 once a sequence has been read in
  int length;

  //open the file
  if((fastaFile = fopen(filename, "r")) == NULL) {
  	return -1;
  }

  sequence[0] = '\0';

  //read the full first sequence, discarding unnecessary headers
  readFlag=0;
  length = 0;
  while(fgets(input,256,fastaFile)!=NULL) {
    //is it a header or a comment?
  	if(input[0]=='>' || input[0]==';') {
  		if(readFlag) break;
  		else continue;
  	}
  	else readFlag = 1;

  	cleanString(input);
  	length += strlen(input);

  	strncat(sequence,input,MAX_SEQUENCE_LENGTH-length - 1);
  }
  //Add a terminatng null character, just in case
  sequence[length] = '\0';

  fclose(fastaFile);
  return length;
}


/****************************************/
/*Implementation of Levenshtein distance*/
/****************************************/

__global__ void levenshtein_distance(char *s,char*t, int one, int two, int *d)
/*Compute levenshtein distance between s and t*/
{
  	//Step 1
	int k,i,j,cost;
	int distance = 0;
	if(one!=0&&two!=0)
	{
		two++;
		one++;
    	//Step 2	
		for(k=0;k<one;k++){
			d[k]=k;
		}
		for(k=0;k<two;k++){
			d[k*one]=k;
		}
    	//Step 3 and 4	
		for(i=1;i<one;i++){
			for(j=1;j<two;j++)
			{
        		//Step 5
				if(s[i-1]==t[j-1]){
					cost=0;
				}
				else{
					cost=1;
				}
        		//Step 6
        		int min = d[(j-1)*one+i]+1;
        		if (d[j*one+i-1]+1 < min){
        			min = d[j*one+i-1]+1;
        		}
        		if (d[(j-1)*one+i-1]+cost < min){
        			min = d[(j-1)*one+i-1]+cost;
        		}
        		d[j*one+i] = min;		 
			}
		}
		printf("%d\n", distance);
	}
		else
			printf ("-1");
}

int main(int argc, char *argv[]) {
	char A[MAX_SEQUENCE_LENGTH+1];
	char B[MAX_SEQUENCE_LENGTH+1];
	if(argc < 3) {
		printf("Usage: new_edit_distance <sequence1> <sequence2>\n");
		printf("<sequence1>: file containing the first sequence, FASTA format\n");
		printf("<sequence2>: file containing the second sequence, FASTA format\n");
		return EXIT_FAILURE;
	}

	n = importFASTA(argv[1],A);
	m = importFASTA(argv[2],B);

	char *d_A, *d_B;
	int *d_D;                         
	hipMalloc(&d_A, MAX_SEQUENCE_LENGTH+1);
	hipMalloc(&d_B, MAX_SEQUENCE_LENGTH+1);
	hipMalloc(&d_D, (sizeof(int))*(m+1)*(n+1));

	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(n/threadsPerBlock.x);

	levenshtein_distance<<<1, 1>>>(d_A,d_B, n, m, d_D);
	hipDeviceSynchronize();
	printf ("%s\n", hipGetErrorString(hipGetLastError()));

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_D);

	return EXIT_SUCCESS;
}
