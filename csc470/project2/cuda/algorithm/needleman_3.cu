
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>
#include <ctype.h>

#define MAX_SEQUENCE_LENGTH 100000

int  n; 
int  m;
int levenshtein_distance(char *s,char*t);
int minimum(int a,int b,int c);

//-----------------------------------------------------------------------------
void cleanString(char string[]) {
  //Removes all spaces from string pointed to by "string", converts characters
  //to uppercase, and deletes a terminating newline character.
	int i, current;
	int length = strlen(string);

	current = 0;
	for(i=0;i<length;i++) {
		if(string[i]=='\n') {
			string[current++] = '\0';
			break;
		}
		else if(string[i]!=' ') {
			string[current++] = toupper(string[i]);
		}
	}
}
//-----------------------------------------------------------------------------
int importFASTA(char *filename, char *sequence) {
  //Reads a file, located at path specified by "filename", containing a FASTA
  //sequence. It finds the first full, complete sequence in the file, stores
  //it in "sequence", and returns the length of the sequence, or -1 on failure.
	FILE *fastaFile;
	char input[256];
  int readFlag; //set to 1 once a sequence has been read in
  int length;

  //open the file
  if((fastaFile = fopen(filename, "r")) == NULL) {
  	return -1;
  }

  sequence[0] = '\0';

  //read the full first sequence, discarding unnecessary headers
  readFlag=0;
  length = 0;
  while(fgets(input,256,fastaFile)!=NULL) {
    //is it a header or a comment?
  	if(input[0]=='>' || input[0]==';') {
  		if(readFlag) break;
  		else continue;
  	}
  	else readFlag = 1;

  	cleanString(input);
  	length += strlen(input);

  	strncat(sequence,input,MAX_SEQUENCE_LENGTH-length - 1);
  }
  //Add a terminatng null character, just in case
  sequence[length] = '\0';

  fclose(fastaFile);
  return length;
}


/****************************************/
/*Implementation of Levenshtein distance*/
/****************************************/

__global__ void levenshtein_distance(char *s,char*t, int one, int two, int *d)
/*Compute levenshtein distance between s and t*/
{
  	//Step 1
	int cost;
	//int distance;
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	int j = (blockIdx.y * blockDim.y) + threadIdx.y;
	if (i>=one || j>=two)
        return;
	if(one!=0&&two!=0){
		
		two++;
		one++;

		if(s[i-1]==t[j-1]){
			cost=0;
		}
		else{
			cost=1;
		}

		int min = d[(j-1)*one+i]+1;
		if (d[j*one+i-1]+1 < min){
			min = d[j*one+i-1]+1;
		}
		if (d[(j-1)*one+i-1]+cost < min){
			min = d[(j-1)*one+i-1]+cost;
		}
		d[j*one+i] = min;
		
	}
	else
		printf ("-1");
}

int main(int argc, char *argv[]) {
	char A[MAX_SEQUENCE_LENGTH+1];
	char B[MAX_SEQUENCE_LENGTH+1];
	if(argc < 3) {
		printf("Usage: new_edit_distance <sequence1> <sequence2>\n");
		printf("<sequence1>: file containing the first sequence, FASTA format\n");
		printf("<sequence2>: file containing the second sequence, FASTA format\n");
		return EXIT_FAILURE;
	}

	n = importFASTA(argv[1],A);
	m = importFASTA(argv[2],B);
	//int k;

	int k, *d, *d_D;
	char *d_A, *d_B;                
	hipMalloc(&d_A, MAX_SEQUENCE_LENGTH+1);
	hipMalloc(&d_B, MAX_SEQUENCE_LENGTH+1);
	hipMalloc(&d_D, (sizeof(int))*(m+1)*(n+1));

	d = (int *)malloc(sizeof(int)*(m+1)*(n+1));

	for(k=0;k<n;k++){
		d[k]=k;
	}
	for(k=0;k<m-1;k++){
		d[k*n]=k;
	}

	hipMemcpy( d_D, d, sizeof(int)*(m+1)*(n+1), hipMemcpyHostToDevice );

	dim3 threadsPerBlock(32, 32);
	dim3 numBlocks(n/threadsPerBlock.x, m/threadsPerBlock.y);

	levenshtein_distance<<<numBlocks, threadsPerBlock>>>(d_A,d_B, n, m, d_D);
	hipDeviceSynchronize();
	//printf ("%s\n", cudaGetErrorString(cudaGetLastError()));

	hipMemcpy( d, d_D, sizeof(int)*(m+1)*(n+1), hipMemcpyDeviceToHost );

	printf("%d\n", d[n*m-1]);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_D);
	free(d);

	return EXIT_SUCCESS;
}
