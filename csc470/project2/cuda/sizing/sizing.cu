
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void multi_thread(void)
{
  printf("Hello, world from the device!\n");
}

int main(void)
{
  // greet from the host
  printf("Hello, world from the host!\n");

  // launch a kernel with a single thread to greet from the device
  multi_thread<<<1,1>>>();
  hipDeviceSynchronize();

  return 0;
}